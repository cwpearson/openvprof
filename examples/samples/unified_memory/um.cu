/*
 * Copyright 2014-2015 NVIDIA Corporation. All rights reserved.
 *
 * Sample CUPTI app to demonstrate the usage of unified memory counter profiling
 *
 */

 #include <stdio.h>
 #include <hip/hip_runtime.h>
 #include <stdlib.h>
 
 #define CUPTI_CALL(call)                                                    \
 do {                                                                        \
     CUptiResult _status = call;                                             \
     if (_status != CUPTI_SUCCESS) {                                         \
       const char *errstr;                                                   \
       cuptiGetResultString(_status, &errstr);                               \
       fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
               __FILE__, __LINE__, #call, errstr);                           \
       exit(-1);                                                             \
     }                                                                       \
 } while (0)
 
 #define DRIVER_API_CALL(apiFuncCall)                                           \
 do {                                                                           \
     hipError_t _status = apiFuncCall;                                            \
     if (_status != hipSuccess) {                                             \
         fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                 __FILE__, __LINE__, #apiFuncCall, _status);                    \
         exit(-1);                                                              \
     }                                                                          \
 } while (0)
 
 #define RUNTIME_API_CALL(apiFuncCall)                                          \
 do {                                                                           \
     hipError_t _status = apiFuncCall;                                         \
     if (_status != hipSuccess) {                                              \
         fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                 __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
         exit(-1);                                                              \
     }                                                                          \
 } while (0)
 

 
 template<class T>
 __host__ __device__ void checkData(const char *loc, T *data, int size, int expectedVal) {
     int i;
 
     for (i = 0; i < size / (int)sizeof(T); i++) {
         if (data[i] != expectedVal) {
             printf("Mismatch found on %s\n", loc);
             printf("Address 0x%p, Observed = 0x%x Expected = 0x%x\n", data+i, data[i], expectedVal);
             break;
         }
     }
 }
 
 template<class T>
 __host__ __device__ void writeData(T *data, int size, int writeVal) {
     int i;
 
     for (i = 0; i < size / (int)sizeof(T); i++) {
         data[i] = writeVal;
     }
 }
 
 __global__ void testKernel(int *data, int size, int expectedVal)
 {
     checkData("GPU", data, size, expectedVal);
     writeData(data, size, -expectedVal);
 }
 
 int main(int argc, char **argv)
 {
     int deviceCount;
     int *data = NULL;
     int size = 64*1024;     // 64 KB
     int i = 123;
 
     DRIVER_API_CALL(hipInit(0));
     DRIVER_API_CALL(hipGetDeviceCount(&deviceCount));
 
     if (deviceCount == 0) {
         printf("There is no device supporting CUDA.\n");
         exit(-1);
     }
 
 
     // allocate unified memory
     printf("Allocation size in bytes %d\n", size);
     RUNTIME_API_CALL(hipMallocManaged(&data, size));
 
     // CPU access
     writeData(data, size, i);
     // kernel launch
     testKernel<<<1,1>>>(data, size, i);
     RUNTIME_API_CALL(hipDeviceSynchronize());
     // CPU access
     checkData("CPU", data, size, -i);
 
     // free unified memory
     RUNTIME_API_CALL(hipFree(data));
 
 
     hipDeviceReset();
 
     return 0;
 }
 